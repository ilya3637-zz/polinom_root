#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <windows.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include<clocale>
#include <cmath>
#include "hip/hip_runtime.h"
#include <vector>
#include <iomanip>
#include <ctime>
#include <algorithm>
#include <set>


// ïðîâåðêà ïðåäïîëîæèòåëüíûõ êîðíåé íà  GPU
__global__ void resultGPU(double* devarrayRR, int temp_n, int* arrayS, int k) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	double tempResult = blockDim.z * blockIdx.z + threadIdx.z;

	for (int i = 0; i < temp_n; i++) {
		tempResult = 0;
		for (int j = 0; j < k; j++) {
			tempResult += arrayS[j] * pow(devarrayRR[i], k - j - 1);
		}
		if (tempResult != 0) devarrayRR[i] = 0;
	}
}

// ïîèñêà äåëèòåëåé p=array[0], q=array[n-1] è çàïèñü èõ â âåêòîð íà GPU
__global__ void pqr(double* devArrayP, double* devArrayQ, double* devArrayR, int p, int q) {

	int i = blockDim.x * blockIdx.x + threadIdx.x; 
	int d = blockDim.y * blockIdx.y + threadIdx.y;
	i = 1;
	devArrayQ[0] = 1;
	if (q < 0) q = q * (-1);
	for (d = (q / 2) + 1; d > 1; d--)
		if (q % d == 0) {
			devArrayQ[i] = d;
			i++;
		}

	i = 1;
	devArrayP[0] = 1;
	if (p < 0) p = p * (-1);
	for (d = (p / 2) + 1; d > 1; d--)
		if (p % d == 0) {
			devArrayP[i] = d;
			i++;
		}

	i = 0;
	for (int ip = 0; ip < 100; ip++) {
		for (int iq = 0; iq < 100; iq++) {	
			if ((devArrayP[ip] != 0) && (devArrayQ[iq] != 0)) {
				devArrayR[i] = devArrayP[ip] / devArrayQ[iq];
				i++;
				devArrayR[i] = -devArrayP[ip] / devArrayQ[iq];
				i++;
			}
		}
	}



}

int main(void) {

	// êîäèðîâêà
	setlocale(0, "");

	// n ñòåïåíü ïîëèíîìà
	std::cout << "Ââåäèòå ìàêñèìàëüíóþ ñòåïåíü ïîëèíîìà: ";
	int n;
	std::cin >> n;
	n++;
	int* arrayS = new int[n];

	// Ââîä ìàññèâà ñ êëàâèàòóðû
	for (int i = 0; i < n; i++)
	{
		std::cin >> arrayS[i];
	}

	unsigned int start_time = clock();
	// Âûâîä F íà ýêðàí
	std::cout << " F = ";
	for (int i = 0; i < n; i++)
	{
		if (arrayS[i] >= 0)
			std::cout << " + " << arrayS[i] << " * x^" << (n - 1 - i) << " ";
		else
			std::cout << arrayS[i] << "*x^" << (n - 1 - i) << " ";
	}
	std::cout << " = 0\n";

	// ïîèñêà äåëèòåëåé p=array[0], q=array[n-1] è çàïèñü èõ â âåêòîðà
	double* arrayPP = new double[100];
	double* arrayQQ = new double[100];
	double* arrayRR = new double[10000];
	for (int i = 0; i < 100; i++) {
		arrayPP[i] = 0;
	}
	for (int i = 0; i < 100; i++) {
		arrayQQ[i] = 0;
	}
	for (int i = 0; i < 1000; i++) {
		arrayRR[i] = 0;
	}
	double* devArrayP;
	double* devArrayQ;
	double* devArrayR;

	hipMalloc((void**)&devArrayP, sizeof(double)*100);
	hipMalloc((void**)&devArrayQ, sizeof(double) * 100);
	hipMalloc((void**)&devArrayR, sizeof(double) * 10000);

	pqr <<< 1, 1 >>>(devArrayP, devArrayQ, devArrayR, arrayS[n - 1], arrayS[0]);
	hipMemcpy(arrayRR, devArrayR, sizeof(double)*10000, hipMemcpyDeviceToHost);
	hipMemcpy(arrayPP, devArrayP, sizeof(double) * 100, hipMemcpyDeviceToHost);
	hipMemcpy(arrayQQ, devArrayQ, sizeof(double) * 100, hipMemcpyDeviceToHost);
	hipFree(devArrayP);
	hipFree(devArrayQ);
	hipFree(devArrayR);


	//âûçîâ ïðîâåðêè íà GPU	
	double* devarrayRR;
	int* devArrayS;
	hipMalloc((void**)&devarrayRR, sizeof(double)*10000);
	hipMalloc((void**)&devArrayS, n * sizeof(int));
	hipMemcpy(devarrayRR, arrayRR, sizeof(double)*10000, hipMemcpyHostToDevice);
	hipMemcpy(devArrayS, arrayS, sizeof(int)*n, hipMemcpyHostToDevice);

	resultGPU <<< 100, 100 >>>(devarrayRR, 10000, devArrayS, n);

	hipMemcpy(arrayRR, devarrayRR, sizeof(double)*10000, hipMemcpyDeviceToHost);
	hipFree(devarrayRR);



	//õðåíü----------------------------------------------------------------------------------------------
	std::cout << "Äåëèòåëè p:  ";
	for (int i = 0; i < 100; i++) {
		if (arrayPP[i] != 0) std::cout << arrayPP[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "Äåëèòåëè q:  ";
	for (int i = 0; i < 100; i++) {
		if (arrayQQ[i] != 0) std::cout << arrayQQ[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "Ïðåäïîëàãàåìûå êîðíè:  ";
	for (int i = 0; i < 10000; i++) {
		if (arrayRR[i] != NULL) std::cout << arrayRR[i] << "   ";
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "Âåðíûå êîðíè GPU:  ";
	for (int i = 0; i < 10000; i++) {
		if (arrayRR[i] != 0) std::cout << arrayRR[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";
	//êîíåö õðåíè----------------------------------------------------------------------------------------------



	// âðåìÿ èñïîëíåíèÿ
	unsigned int end_time = clock();
	unsigned int search_time = end_time - start_time;
	std::cout << "Âðåìÿ ðàáîòû îáùåå " << search_time << " ìèëèñåêóíä \n";
	system("pause");
	return 0;
}
