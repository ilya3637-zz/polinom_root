#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <windows.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include<clocale>
#include <cmath>
#include "hip/hip_runtime.h"
#include <vector>
#include <iomanip>
#include <ctime>
#include <algorithm>
#include <set>



// ïðîâåðêà ïðåäïîëîæèòåëüíûõ êîðíåé íà  CPU
std::vector<double> result(std::vector<double> arrayR_F, int* array_F, int k) {
	std::vector<double> arrayResult;
	int n = arrayR_F.size();	

	for (int i = 0; i < n; i++) { //ïðîõîä ïî ïðåäïîëàãàåìûì êîðíÿì
		double tempResult = 0;
		for (int j = 0; j < k; j++) { //ïðîõîä ïî ñòåïåíÿì (ýëåìåíòàì) ïîëèíîìà
			tempResult += array_F[j] * pow(arrayR_F[i], k - j - 1);
		}
		if (tempResult == 0) arrayResult.push_back(arrayR_F[i]);
	}
	return arrayResult;
}





int main(void) {
	
	// êîäèðîâêà
	setlocale(0, "");

	// n ñòåïåíü ïîëèíîìà
	std::cout<< "Ââåäèòå ìàêñèìàëüíóþ ñòåïåíü ïîëèíîìà: ";
	int n;
	std::cin >> n;
	n++;
	int* arrayS = new int[n];

	// Ââîä ìàññèâà ñ êëàâèàòóðû
	for (int i = 0; i < n; i++)
	{
		std::cin >> arrayS[i];
	}

	unsigned int start_time = clock();
	// Âûâîä F íà ýêðàí
	std::cout << " F = ";
	for (int i = 0; i < n; i++)
	{
		if (arrayS[i] >= 0) 
			std::cout << " + " << arrayS[i] << " * x^" << (n - 1 - i) << " ";
		else
			std::cout << arrayS[i] << "*x^" << (n - 1 - i) << " ";
	}
	std::cout << " = 0\n";

	// ïîèñêà äåëèòåëåé p=array[0], q=array[n-1] è çàïèñü èõ â âåêòîðà
	std::vector<double> arrayP(1, arrayS[n-1]);
	std::vector<double> arrayQ(1, arrayS[0]);

	for (int d = (fabs(arrayS[0]) / 2) + 1; d > 1; d--)
		if (arrayS[0] % d == 0)
			arrayQ.push_back(d);

	for (int d = (fabs(arrayS[n - 1]) / 2) + 1; d > 1; d--)
		if (arrayS[n - 1] % d == 0)
			arrayP.push_back(d);

	arrayP.push_back(1);
	arrayQ.push_back(1);

	// ñîñòàâëåíèå âåêòîðà arrayR ïðåäïîëîæèòåëüíûõ êîðíåé (êîìáèíàöèé áåç ïîâòîðåíèé) èç âñåõ +- p/q 
	std::vector<double> arrayR;
	for (int ip = 0; ip < arrayP.size(); ip++){
		for (int iq = 0; iq < arrayQ.size(); iq++) {
			arrayR.push_back(double(arrayP[ip] / arrayQ[iq]));
			arrayR.push_back(double(-arrayP[ip] / arrayQ[iq]));
		}
	}

	// ñîðòèðîâêà è óáîðêà äóáëèêàòîâ èç arrayR
	std::sort(arrayR.begin(), arrayR.end());
	arrayR.erase(std::unique(arrayR.begin(), arrayR.end()), arrayR.end());

	//âûçîâ ïðîâåðêè íà CPU
	
	std::vector<double> arrayResult1;
	arrayResult1 = result(arrayR, arrayS, n);
	

	//õðåíü----------------------------------------------------------------------------------------------
	std::cout << "Äåëèòåëè p:  ";
	for (int i = 0; i < arrayP.size(); i++) {
		std::cout << arrayP[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "Äåëèòåëè q:  ";
	for (int i = 0; i < arrayQ.size(); i++) {
		std::cout << arrayQ[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";
	
	std::cout << "Ïðåäïîëàãàåìûå êîðíè:  ";
	for (int i = 0; i < arrayR.size(); i++) {
		std::cout << arrayR[i] << "   ";
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "Âåðíûå êîðíè CPU:  ";
	for (int i = 0; i < arrayResult1.size(); i++) {
		std::cout << arrayResult1[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";
	//êîíåö õðåíè----------------------------------------------------------------------------------------------


	
	// âðåìÿ èñïîëíåíèÿ
	unsigned int end_time = clock();
	unsigned int search_time = end_time - start_time;
	std::cout << "Âðåìÿ ðàáîòû îáùåå " << search_time << " ìèëèñåêóíä \n";
	system("pause");
	return 0;
}
