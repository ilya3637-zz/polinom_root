#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <windows.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include<clocale>
#include <cmath>
#include "hip/hip_runtime.h"
#include <vector>
#include <iomanip>
#include <ctime>
#include <algorithm>
#include <set>



// �������� ����������������� ������ ��  CPU
std::vector<double> result(std::vector<double> arrayR_F, int* array_F, int k) {
	std::vector<double> arrayResult;
	int n = arrayR_F.size();	

	for (int i = 0; i < n; i++) { //������ �� �������������� ������
		double tempResult = 0;
		for (int j = 0; j < k; j++) { //������ �� �������� (���������) ��������
			tempResult += array_F[j] * pow(arrayR_F[i], k - j - 1);
		}
		if (tempResult == 0) arrayResult.push_back(arrayR_F[i]);
	}
	return arrayResult;
}





int main(void) {
	
	// ���������
	setlocale(0, "");

	// n ������� ��������
	std::cout<< "������� ������������ ������� ��������: ";
	int n;
	std::cin >> n;
	n++;
	int* arrayS = new int[n];

	// ���� ������� � ����������
	for (int i = 0; i < n; i++)
	{
		std::cin >> arrayS[i];
	}

	unsigned int start_time = clock();
	// ����� F �� �����
	std::cout << " F = ";
	for (int i = 0; i < n; i++)
	{
		if (arrayS[i] >= 0) 
			std::cout << " + " << arrayS[i] << " * x^" << (n - 1 - i) << " ";
		else
			std::cout << arrayS[i] << "*x^" << (n - 1 - i) << " ";
	}
	std::cout << " = 0\n";

	// ������ ��������� p=array[0], q=array[n-1] � ������ �� � �������
	std::vector<double> arrayP(1, arrayS[n-1]);
	std::vector<double> arrayQ(1, arrayS[0]);

	for (int d = (fabs(arrayS[0]) / 2) + 1; d > 1; d--)
		if (arrayS[0] % d == 0)
			arrayQ.push_back(d);

	for (int d = (fabs(arrayS[n - 1]) / 2) + 1; d > 1; d--)
		if (arrayS[n - 1] % d == 0)
			arrayP.push_back(d);

	arrayP.push_back(1);
	arrayQ.push_back(1);

	// ����������� ������� arrayR ����������������� ������ (���������� ��� ����������) �� ���� +- p/q 
	std::vector<double> arrayR;
	for (int ip = 0; ip < arrayP.size(); ip++){
		for (int iq = 0; iq < arrayQ.size(); iq++) {
			arrayR.push_back(double(arrayP[ip] / arrayQ[iq]));
			arrayR.push_back(double(-arrayP[ip] / arrayQ[iq]));
		}
	}

	// ���������� � ������ ���������� �� arrayR
	std::sort(arrayR.begin(), arrayR.end());
	arrayR.erase(std::unique(arrayR.begin(), arrayR.end()), arrayR.end());

	//����� �������� �� CPU
	
	std::vector<double> arrayResult1;
	arrayResult1 = result(arrayR, arrayS, n);
	

	//�����----------------------------------------------------------------------------------------------
	std::cout << "�������� p:  ";
	for (int i = 0; i < arrayP.size(); i++) {
		std::cout << arrayP[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "�������� q:  ";
	for (int i = 0; i < arrayQ.size(); i++) {
		std::cout << arrayQ[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";
	
	std::cout << "�������������� �����:  ";
	for (int i = 0; i < arrayR.size(); i++) {
		std::cout << arrayR[i] << "   ";
	}
	std::cout << " \n";
	std::cout << " \n";

	std::cout << "������ ����� CPU:  ";
	for (int i = 0; i < arrayResult1.size(); i++) {
		std::cout << arrayResult1[i] << ' ';
	}
	std::cout << " \n";
	std::cout << " \n";
	//����� �����----------------------------------------------------------------------------------------------


	
	// ����� ����������
	unsigned int end_time = clock();
	unsigned int search_time = end_time - start_time;
	std::cout << "����� ������ ����� " << search_time << " ���������� \n";
	system("pause");
	return 0;
}